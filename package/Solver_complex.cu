//#include "cuSolverDn_AtA.cu"
#include "Solver_manager.hh"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <assert.h>
#include <cmath> 
#include <hip/hip_runtime.h>
#include "SI.h"
#include "hipblas.h"
#include "hipsolver.h"
#include "hip/hip_runtime_api.h"
#include "helper_cusolver.h"

/*
 *  solve A*x = b by QR
 *
 */
int linearSolverQR(
    hipsolverHandle_t handle,
    int n,
    const float *Acopy,
    int lda,
    const float *b,
    float *x)
{
    hipblasHandle_t cublasHandle = NULL; // used in residual evaluation
    int bufferSize = 0;
    int bufferSize_geqrf = 0;
    int bufferSize_ormqr = 0;
    int *info = NULL;
    float *buffer = NULL;
    float *A = NULL;
    float *tau = NULL;
    int h_info = 0;
    float start, stop;
    float time_solve;
    const float one = 1.0;

    checkCudaErrors(hipblasCreate(&cublasHandle));

    checkCudaErrors(hipsolverDnSgeqrf_bufferSize(handle, n, n, (float*)Acopy, lda, &bufferSize_geqrf));
    checkCudaErrors(hipsolverDnSormqr_bufferSize(
        handle,
        HIPBLAS_SIDE_LEFT,
        HIPBLAS_OP_T,
        n,
        1,
        n,
        A,
        lda,
        NULL,
        x,
        n,
        &bufferSize_ormqr));

    printf("buffer_geqrf = %d, buffer_ormqr = %d \n", bufferSize_geqrf, bufferSize_ormqr);
    
    bufferSize = (bufferSize_geqrf > bufferSize_ormqr)? bufferSize_geqrf : bufferSize_ormqr ; 

    checkCudaErrors(hipMalloc(&info, sizeof(int)));
    checkCudaErrors(hipMalloc(&buffer, sizeof(float)*bufferSize));
    checkCudaErrors(hipMalloc(&A, sizeof(float)*lda*n));
    checkCudaErrors(hipMalloc ((void**)&tau, sizeof(float)*n));

// prepare a copy of A because getrf will overwrite A with L
    checkCudaErrors(hipMemcpy(A, Acopy, sizeof(float)*lda*n, hipMemcpyDeviceToDevice));

    checkCudaErrors(hipMemset(info, 0, sizeof(int)));

    start = second();
    start = second();

// compute QR factorization
    checkCudaErrors(hipsolverDnSgeqrf(handle, n, n, A, lda, tau, buffer, bufferSize, info));

    checkCudaErrors(hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost));

    if ( 0 != h_info ){
        fprintf(stderr, "Error: QR factorization failed, check %d parameter\n", h_info);
    }

    checkCudaErrors(hipMemcpy(x, b, sizeof(float)*n, hipMemcpyDeviceToDevice));

    // compute Q^T*b
    checkCudaErrors(hipsolverDnSormqr(
        handle,
        HIPBLAS_SIDE_LEFT,
        HIPBLAS_OP_T,
        n,
        1,
        n,
        A,
        lda,
        tau,
        x,
        n,
        buffer,
        bufferSize,
        info));

    // x = R \ Q^T*b
    checkCudaErrors(hipblasStrsm(
         cublasHandle,
         HIPBLAS_SIDE_LEFT,
         HIPBLAS_FILL_MODE_UPPER,
         HIPBLAS_OP_N,
         HIPBLAS_DIAG_NON_UNIT,
         n,
         1,
         &one,
         A,
         lda,
         x,
         n));
    checkCudaErrors(hipDeviceSynchronize());
    stop = second();

    time_solve = stop - start;
    fprintf (stdout, "timing: QR = %10.6f sec\n", time_solve);

    if (cublasHandle) { checkCudaErrors(hipblasDestroy(cublasHandle)); }
    if (info  ) { checkCudaErrors(hipFree(info  )); }
    if (buffer) { checkCudaErrors(hipFree(buffer)); }
    if (A     ) { checkCudaErrors(hipFree(A)); }
    if (tau   ) { checkCudaErrors(hipFree(tau)); }

    return 0;
}


DnSolver::DnSolver (float* array_host_, float* rhs_, int rows_, int cols_) 
{
    h_A = array_host_;
    rowsA = rows_;
    colsA = cols_;
    lda = rows_;
    h_b = rhs_;


    checkCudaErrors(hipsolverDnCreate(&handle));
    checkCudaErrors(hipblasCreate(&cublasHandle));
    checkCudaErrors(hipStreamCreate(&stream));

    checkCudaErrors(hipsolverSetStream(handle, stream));
    checkCudaErrors(hipblasSetStream(cublasHandle, stream));


    checkCudaErrors(hipMalloc((void **)&d_A, sizeof(float)*lda*colsA));
    checkCudaErrors(hipMalloc((void **)&d_x, sizeof(float)*colsA));
    checkCudaErrors(hipMalloc((void **)&d_b, sizeof(float)*rowsA));
    // checkCudaErrors(hipMalloc((void **)&d_r, sizeof(float)*rowsA));
    // checkCudaErrors(hipMalloc((void **)&d_tr, sizeof(float)*rowsA));

    printf("prepare data on device\n");
    checkCudaErrors(hipMemcpy(d_A, h_A, sizeof(float)*lda*colsA, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_b, h_b, sizeof(float)*rowsA, hipMemcpyHostToDevice));
}

void DnSolver::solve() {
    printf("step 6: compute AtA \n");
    hipblasStatus_t cbstat;
    float al =1.0;// al =1
    float bet =0.0;// bet =0
    //float* dAcopy;
    float* dAtA;
    //checkCudaErrors(hipMalloc(&dAcopy, sizeof(float)*lda*colsA));
    checkCudaErrors(hipMalloc(&dAtA, sizeof(float)*colsA*colsA));
    //checkCudaErrors(hipMemcpy(dAcopy, d_A, sizeof(float)*lda*colsA, hipMemcpyDeviceToDevice));
    //cbstat = hipblasDgemm(cublasHandle,HIPBLAS_OP_T,HIPBLAS_OP_N,colsA,rowsA,rowsA,&al,d_A,colsA,d_A,rowsA,&bet,dAtA,colsA);
    cbstat = hipblasSgemm(cublasHandle,HIPBLAS_OP_T,HIPBLAS_OP_N,colsA,colsA,rowsA,&al,d_A,rowsA,d_A,rowsA,&bet,dAtA,colsA);

    printf("step 7: compute At*b \n");
    float* d_Atb;
    checkCudaErrors(hipMalloc((void **)&d_Atb, sizeof(float)*colsA));
    cbstat = hipblasSgemv(cublasHandle,HIPBLAS_OP_T,colsA,colsA,&al,d_A,colsA,d_b,1,&bet,d_Atb,1);

    if (cublasHandle) { checkCudaErrors(hipblasDestroy(cublasHandle)); }
    checkCudaErrors(hipblasCreate(&cublasHandle));
    checkCudaErrors(hipblasSetStream(cublasHandle, stream));
    printf("step 8: solves AtA*x = At*b \n");

    linearSolverQR(handle, colsA, dAtA, colsA, d_Atb, d_x);

}

void DnSolver::retrieve_to(float* h_x)
{
    checkCudaErrors(hipMemcpy(h_x, d_x, sizeof(float)*colsA, hipMemcpyDeviceToHost));
    printf("x0 = %E \n", h_x[0]);
}

DnSolver::~DnSolver()
{
    if (handle) { checkCudaErrors(hipsolverDnDestroy(handle)); }
    if (cublasHandle) { checkCudaErrors(hipblasDestroy(cublasHandle)); }
    if (stream) { checkCudaErrors(hipStreamDestroy(stream)); }

    if (h_A) { free(h_A); }
    if (h_x) { free(h_x); }
    if (h_b) { free(h_b); }

    if (d_A) { checkCudaErrors(hipFree(d_A)); }
    if (d_x) { checkCudaErrors(hipFree(d_x)); }
    if (d_b) { checkCudaErrors(hipFree(d_b)); }
}
