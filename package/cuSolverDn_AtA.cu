

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <assert.h>
#include <cmath> 
#include <hip/hip_runtime.h>
#include "SI.h"
#include "hipblas.h"
#include "hipsolver.h"
#include "hip/hip_runtime_api.h"

#include "helper_cusolver.h"


template <typename T_ELEM>
int loadMMSparseMatrix(
    char *filename,
    char elem_type,
    bool csrFormat,
    int *m,
    int *n,
    int *nnz,
    T_ELEM **aVal,
    int **aRowInd,
    int **aColInd,
    int extendSymMatrix);

void UsageDN(void)
{
    printf( "<options>\n");
    printf( "-h          : display this help\n");
    printf( "-R=<name>    : choose a linear solver\n");
    printf( "              chol (cholesky factorization), this is default\n");
    printf( "              qr   (QR factorization)\n");
    printf( "              lu   (LU factorization)\n");
    printf( "-lda=<int> : leading dimension of A , m by default\n");
    printf( "-file=<filename>: filename containing a matrix in MM format\n");
    printf( "-device=<device_id> : <device_id> if want to run on specific GPU\n");

    exit( 0 );
}
/*
 *  solve A*x = b by Cholesky factorization
 *
 */
int linearSolverCHOL(
    hipsolverHandle_t handle,
    int n,
    const float *Acopy,
    int lda,
    const float *b,
    float *x)
{
    int bufferSize = 0;
    int *info = NULL;
    float *buffer = NULL;
    float *A = NULL;
    int h_info = 0;
    float start, stop;
    float time_solve;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    checkCudaErrors(hipsolverDnSpotrf_bufferSize(handle, uplo, n, (float*)Acopy, lda, &bufferSize));

    checkCudaErrors(hipMalloc(&info, sizeof(int)));
    checkCudaErrors(hipMalloc(&buffer, sizeof(float)*bufferSize));
    checkCudaErrors(hipMalloc(&A, sizeof(float)*lda*n));


    // prepare a copy of A because potrf will overwrite A with L
    checkCudaErrors(hipMemcpy(A, Acopy, sizeof(float)*lda*n, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemset(info, 0, sizeof(int)));

    start = second();
    start = second();

    checkCudaErrors(hipsolverDnSpotrf(handle, uplo, n, A, lda, buffer, bufferSize, info));

    checkCudaErrors(hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost));

    if ( 0 != h_info ){
        fprintf(stderr, "Error: Cholesky factorization failed, check %d parameter\n", h_info);
    }

    checkCudaErrors(hipMemcpy(x, b, sizeof(float)*n, hipMemcpyDeviceToDevice));

    checkCudaErrors(hipsolverDnSpotrs(handle, uplo, n, 1, A, lda, x, n, info));

    checkCudaErrors(hipDeviceSynchronize());
    stop = second();

    time_solve = stop - start;
    fprintf (stdout, "timing: cholesky = %10.6f sec\n", time_solve);

    if (info  ) { checkCudaErrors(hipFree(info)); }
    if (buffer) { checkCudaErrors(hipFree(buffer)); }
    if (A     ) { checkCudaErrors(hipFree(A)); }

    return 0;
}


/*
 *  solve A*x = b by LU with partial pivoting
 *
 */
int linearSolverLU(
    hipsolverHandle_t handle,
    int n,
    const float *Acopy,
    int lda,
    const float *b,
    float *x)
{
    int bufferSize = 0;
    int *info = NULL;
    float *buffer = NULL;
    float *A = NULL;
    int *ipiv = NULL; // pivoting sequence
    int h_info = 0;
    float start, stop;
    float time_solve;

    checkCudaErrors(hipsolverDnSgetrf_bufferSize(handle, n, n, (float*)Acopy, lda, &bufferSize));

    checkCudaErrors(hipMalloc(&info, sizeof(int)));
    checkCudaErrors(hipMalloc(&buffer, sizeof(float)*bufferSize));
    checkCudaErrors(hipMalloc(&A, sizeof(float)*lda*n));
    checkCudaErrors(hipMalloc(&ipiv, sizeof(int)*n));


    // prepare a copy of A because getrf will overwrite A with L
    checkCudaErrors(hipMemcpy(A, Acopy, sizeof(float)*lda*n, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemset(info, 0, sizeof(int)));

    start = second();
    start = second();

    checkCudaErrors(hipsolverDnSgetrf(handle, n, n, A, lda, buffer, ipiv, info));
    checkCudaErrors(hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost));

    if ( 0 != h_info ){
        fprintf(stderr, "Error: LU factorization failed, check %d parameter\n", h_info);
    }

    checkCudaErrors(hipMemcpy(x, b, sizeof(float)*n, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipsolverDnSgetrs(handle, HIPBLAS_OP_N, n, 1, A, lda, ipiv, x, n, info));
    checkCudaErrors(hipDeviceSynchronize());
    stop = second();

    time_solve = stop - start;
    fprintf (stdout, "timing: LU = %10.6f sec\n", time_solve);

    if (info  ) { checkCudaErrors(hipFree(info  )); }
    if (buffer) { checkCudaErrors(hipFree(buffer)); }
    if (A     ) { checkCudaErrors(hipFree(A)); }
    if (ipiv  ) { checkCudaErrors(hipFree(ipiv));}

    return 0;
}


int linearSolverSVD(
    hipsolverHandle_t handle, 
    int n,
    const float *Acopy,
    int lda,
    const float *bcopy,
    float *x)
{
    hipblasHandle_t cublasHandle = NULL; // used in residual evaluation
    int m = lda;
    int bufferSize = 0;
    int *info = NULL;
    int h_info = 0;
    float start, stop;
    float time_solve;
    const float one = 1.0;

    // float U[lda*m]; // m-by-m unitary matrix 
    // float VT[lda*n]; // n-by-n unitary matrix
    // float S[n]; //singular value 
    float *d_A = NULL; float *d_SI = NULL; 
    float *d_b = NULL; float *d_S = NULL; 
    float *d_U = NULL; float *d_VT = NULL; 
    float *d_work = NULL; 
    float *d_rwork = NULL; 
    float *d_W = NULL; 
    signed char jobu = 'A'; // all m columns of U 
    signed char jobvt = 'A'; // all n columns of VT 
    // step 1: create cusolverDn/cublas handle 
    checkCudaErrors(hipblasCreate(&cublasHandle)); 

    checkCudaErrors(hipMalloc((void**)&d_A , sizeof(float)*lda*n)); \
    checkCudaErrors(hipMalloc((void**)&d_b , sizeof(float)*m)); 
    checkCudaErrors(hipMalloc((void**)&d_S , sizeof(float)*n)); 
    checkCudaErrors(hipMalloc((void**)&d_SI , sizeof(float)*lda*n)); 
    checkCudaErrors(hipMalloc((void**)&d_U , sizeof(float)*lda*m)); 
    checkCudaErrors(hipMalloc((void**)&d_VT , sizeof(float)*lda*n)); 
    checkCudaErrors(hipMalloc((void**)&info, sizeof(int))); 
    checkCudaErrors(hipMalloc((void**)&d_W , sizeof(float)*lda*n));
    checkCudaErrors(hipMemcpy(d_A, Acopy, sizeof(float)*lda*n, hipMemcpyDeviceToDevice)); //gesvd destroys d_A on exit
    checkCudaErrors(hipMemcpy(d_b, bcopy, sizeof(float)*m, hipMemcpyDeviceToDevice));
    
    // checkMatrix(m, n, d_SI, lda, "zero_SI");
    // checkMatrix(m, n , d_A, lda, "SVD_AtA");
    // checkArray(d_b, m, "SVD_Atb");
    checkCudaErrors(hipsolverDnSgesvd_bufferSize( handle, m, n, &bufferSize ));
    checkCudaErrors(hipMalloc((void**)&d_work , sizeof(float)*bufferSize));

    start = second();

    checkCudaErrors(hipsolverDnSgesvd( 
        handle, jobu, jobvt, m, n, d_A, lda, d_S, d_U, lda, d_VT, lda, d_work, bufferSize, d_rwork, info));
    //checkCudaErrors(hipDeviceSynchronize());
    
    // checkCudaErrors(hipMemcpy(U , d_U , sizeof(float)*lda*m, hipMemcpyDeviceToHost)); 
    // checkCudaErrors(hipMemcpy(VT, d_VT, sizeof(float)*lda*n, hipMemcpyDeviceToHost)); 
    // checkCudaErrors(hipMemcpy(S , d_S , sizeof(float)*n , hipMemcpyDeviceToHost)); 
    checkCudaErrors(hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost));

    if ( 0 != h_info ){
        fprintf(stderr, "Error: SVD failed, check %d parameter\n", h_info);
    }

    // int BLOCK_DIM_X = 32; int BLOCK_DIM_Y = 32;
    // dim3 blockDim(BLOCK_DIM_X, BLOCK_DIM_Y);  
    // dim3 gridDim((n + BLOCK_DIM_X - 1) / BLOCK_DIM_X, (m + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y);
    // initSIGPU<<<gridDim, blockDim>>>(d_SI, d_S, m, n);
    float epsilon = 1.e-9;
    printf("epsilon = %f \n", epsilon);
    initSI<float>(d_SI, d_S, m, n, epsilon, 256);
    //int initStat = initSICPU(d_SI, d_S, m, n, epsilon);
    // U*S*V*x=b; x = VT*Si*UT*b
    // checkMatrix(m, n, d_SI, lda, "SVD_SI");
    // checkArray(d_S, n, "dS");
    // checkMatrix(m, m, d_U, lda, "SVD_U");
    // checkMatrix(n, n, d_VT, lda, "SVD_VT");
    float al = 1.0;// al =1
    float bet = 0.0;// bet =0
    // checkArray(d_b, n, "db");
    checkCudaErrors(hipblasSgemv(cublasHandle,HIPBLAS_OP_T, m, m, &al,d_U, m, d_b,1,&bet,d_b,1));
    // checkArray(d_b, n, "dUtb");
    checkCudaErrors(hipblasSgemv(cublasHandle,HIPBLAS_OP_N, m, n, &al,d_SI, m, d_b,1,&bet,d_b,1));
    // checkArray(d_b, n, "dSiUtb");
    checkCudaErrors(hipblasSgemv(cublasHandle,HIPBLAS_OP_T, n, n, &al,d_VT, n, d_b, 1,&bet,x,1));
    checkCudaErrors(hipDeviceSynchronize());
    stop = second();
    time_solve = stop - start; 
    fprintf (stdout, "timing: SVD = %10.6f sec\n", time_solve);
    // checkArray(x, 20, "d_x");

    if (d_A ) hipFree(d_A); 
    if (d_S ) hipFree(d_S); 
    if (d_SI ) hipFree(d_SI);
    if (d_U ) hipFree(d_U); 
    if (d_VT ) hipFree(d_VT); 
    if (info) hipFree(info); 
    if (d_work ) hipFree(d_work); 
    if (d_rwork) hipFree(d_rwork); 
    if (d_W ) hipFree(d_W); 
    if (cublasHandle ) hipblasDestroy(cublasHandle); 
    // if (cusolverH) hipsolverDnDestroy(cusolverH); 
    return 0;


}
/*
 *  solve A*x = b by QR
 *
 */
int linearSolverQR(
    hipsolverHandle_t handle,
    int n,
    const float *Acopy,
    int lda,
    const float *b,
    float *x)
{
    hipblasHandle_t cublasHandle = NULL; // used in residual evaluation
    int bufferSize = 0;
    int bufferSize_geqrf = 0;
    int bufferSize_ormqr = 0;
    int *info = NULL;
    float *buffer = NULL;
    float *A = NULL;
    float *tau = NULL;
    int h_info = 0;
    float start, stop;
    float time_solve;
    const float one = 1.0;

    checkCudaErrors(hipblasCreate(&cublasHandle));

    checkCudaErrors(hipsolverDnSgeqrf_bufferSize(handle, n, n, (float*)Acopy, lda, &bufferSize_geqrf));
    checkCudaErrors(hipsolverDnSormqr_bufferSize(
        handle,
        HIPBLAS_SIDE_LEFT,
        HIPBLAS_OP_T,
        n,
        1,
        n,
        A,
        lda,
        NULL,
        x,
        n,
        &bufferSize_ormqr));

    printf("buffer_geqrf = %d, buffer_ormqr = %d \n", bufferSize_geqrf, bufferSize_ormqr);
    
    bufferSize = (bufferSize_geqrf > bufferSize_ormqr)? bufferSize_geqrf : bufferSize_ormqr ; 

    checkCudaErrors(hipMalloc(&info, sizeof(int)));
    checkCudaErrors(hipMalloc(&buffer, sizeof(float)*bufferSize));
    checkCudaErrors(hipMalloc(&A, sizeof(float)*lda*n));
    checkCudaErrors(hipMalloc ((void**)&tau, sizeof(float)*n));

// prepare a copy of A because getrf will overwrite A with L
    checkCudaErrors(hipMemcpy(A, Acopy, sizeof(float)*lda*n, hipMemcpyDeviceToDevice));

    checkCudaErrors(hipMemset(info, 0, sizeof(int)));

    start = second();
    start = second();

    // BENCHMARKING: 
    // for (int i=0; i< 1000; i++) {
    //         // compute QR factorization
    //     checkCudaErrors(hipMemcpy(A, Acopy, sizeof(float)*lda*n, hipMemcpyDeviceToDevice));
    //     checkCudaErrors(hipsolverDnSgeqrf(handle, n, n, A, lda, tau, buffer, bufferSize, info));

    //     checkCudaErrors(hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost));

    //     if ( 0 != h_info ){
    //         fprintf(stderr, "Error: QR factorization failed, check %d parameter\n", h_info);
    //     }

    //     checkCudaErrors(hipMemcpy(x, b, sizeof(float)*n, hipMemcpyDeviceToDevice));

    //     // compute Q^T*b
    //     checkCudaErrors(hipsolverDnSormqr(
    //         handle,
    //         HIPBLAS_SIDE_LEFT,
    //         HIPBLAS_OP_T,
    //         n,
    //         1,
    //         n,
    //         A,
    //         lda,
    //         tau,
    //         x,
    //         n,
    //         buffer,
    //         bufferSize,
    //         info));

    //     // x = R \ Q^T*b
    //     checkCudaErrors(hipblasStrsm(
    //          cublasHandle,
    //          HIPBLAS_SIDE_LEFT,
    //          HIPBLAS_FILL_MODE_UPPER,
    //          HIPBLAS_OP_N,
    //          HIPBLAS_DIAG_NON_UNIT,
    //          n,
    //          1,
    //          &one,
    //          A,
    //          lda,
    //          x,
    //          n));
    //     checkCudaErrors(hipDeviceSynchronize());
    // }
// compute QR factorization
    checkCudaErrors(hipsolverDnSgeqrf(handle, n, n, A, lda, tau, buffer, bufferSize, info));

    checkCudaErrors(hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost));

    if ( 0 != h_info ){
        fprintf(stderr, "Error: QR factorization failed, check %d parameter\n", h_info);
    }

    checkCudaErrors(hipMemcpy(x, b, sizeof(float)*n, hipMemcpyDeviceToDevice));

    // compute Q^T*b
    checkCudaErrors(hipsolverDnSormqr(
        handle,
        HIPBLAS_SIDE_LEFT,
        HIPBLAS_OP_T,
        n,
        1,
        n,
        A,
        lda,
        tau,
        x,
        n,
        buffer,
        bufferSize,
        info));

    // x = R \ Q^T*b
    checkCudaErrors(hipblasStrsm(
         cublasHandle,
         HIPBLAS_SIDE_LEFT,
         HIPBLAS_FILL_MODE_UPPER,
         HIPBLAS_OP_N,
         HIPBLAS_DIAG_NON_UNIT,
         n,
         1,
         &one,
         A,
         lda,
         x,
         n));
    checkCudaErrors(hipDeviceSynchronize());
    stop = second();

    time_solve = stop - start;
    fprintf (stdout, "timing: QR = %10.6f sec\n", time_solve);

    if (cublasHandle) { checkCudaErrors(hipblasDestroy(cublasHandle)); }
    if (info  ) { checkCudaErrors(hipFree(info  )); }
    if (buffer) { checkCudaErrors(hipFree(buffer)); }
    if (A     ) { checkCudaErrors(hipFree(A)); }
    if (tau   ) { checkCudaErrors(hipFree(tau)); }

    return 0;
}

